#include "hip/hip_runtime.h"
#include<iostream>
#include<cstdlib>
#include<cstdlib>
#include<hip/hip_runtime.h>
#include<highgui.h>
#include<cv.h>

#define Mask_size  3
#define TILE_SIZE  32
#define BLOCK_SIZE 32
__constant__ char M[Mask_size*Mask_size];

using namespace std;
using namespace cv;

//Mantener los valores y no darles salida del dominio

__device__ unsigned char clamp(int value)//__device__ es llamado por el Kernel
{
    if(value < 0)
        value = 0;
    else
        if(value > 255)
            value = 255;
    return  value;
}


//Convolution Kernel con memoria global

__global__ void convolution2DGlobalMemKernel(unsigned char *In,char *Mask, unsigned char *Out,int Mask_Width,int Rowimg,int Colimg)
{

   unsigned int row = blockIdx.y*blockDim.y+threadIdx.y;
   unsigned int col = blockIdx.x*blockDim.x+threadIdx.x;

   int Pvalue = 0;

   int N_start_point_row = row - (Mask_Width/2);
   int N_start_point_col = col - (Mask_Width/2);

   for(int i = 0; i < Mask_Width; i++)
   {
       for(int j = 0; j < Mask_Width; j++ )
       {
        if((N_start_point_col + j >=0 && N_start_point_col + j < Rowimg)
        &&(N_start_point_row + i >=0 && N_start_point_row + i < Colimg))
        {
          Pvalue += In[(N_start_point_row + i)*Rowimg+(N_start_point_col + j)] * Mask[i*Mask_Width+j];
        }
       }
   }

   Out[row*Rowimg+col] = clamp(Pvalue);
}

//Convolution Kernel con memoria constante

__global__ void convolution2DConstantMemKernel(unsigned char *In,unsigned char *Out,int Mask_Width,int Rowimg,int Colimg)
 {
   unsigned int row = blockIdx.y*blockDim.y+threadIdx.y;
   unsigned int col = blockIdx.x*blockDim.x+threadIdx.x;

   int Pvalue = 0;

   int N_start_point_row = row - (Mask_Width/2);
   int N_start_point_col = col - (Mask_Width/2);

   for(int i = 0; i < Mask_Width; i++)
   {
       for(int j = 0; j < Mask_Width; j++ )
       {
         if((N_start_point_col + j >=0 && N_start_point_col + j < Rowimg)
         &&(N_start_point_row + i >=0 && N_start_point_row + i < Colimg))
         {
           Pvalue += In[(N_start_point_row + i)*Rowimg+(N_start_point_col + j)] * M[i*Mask_Width+j];
         }
       }
    }

   Out[row*Rowimg+col] = clamp(Pvalue);
}

//Convolution Kernel con memoria compartida

__global__ void convolution2DSharedMemKernel(unsigned char *imageInput,unsigned char *imageOutput,
 int maskWidth, int width, int height)
{
    __shared__ float N_ds[TILE_SIZE + Mask_size - 1][TILE_SIZE + Mask_size - 1];
    int n = maskWidth/2;
    int dest = threadIdx.y*TILE_SIZE+threadIdx.x, destY = dest / (TILE_SIZE+Mask_size-1), destX = dest % (TILE_SIZE+Mask_size-1),
        srcY = blockIdx.y * TILE_SIZE + destY - n, srcX = blockIdx.x * TILE_SIZE + destX - n,
        src = (srcY * width + srcX);
    if (srcY >= 0 && srcY < height && srcX >= 0 && srcX < width)
        N_ds[destY][destX] = imageInput[src];
    else
        N_ds[destY][destX] = 0;

    // Segundo procesamiento por lotes
    dest = threadIdx.y * TILE_SIZE + threadIdx.x + TILE_SIZE * TILE_SIZE;
    destY = dest /(TILE_SIZE + Mask_size - 1), destX = dest % (TILE_SIZE + Mask_size - 1);
    srcY = blockIdx.y * TILE_SIZE + destY - n;
    srcX = blockIdx.x * TILE_SIZE + destX - n;
    src = (srcY * width + srcX);
    if (destY < TILE_SIZE + Mask_size - 1)
    {
        if (srcY >= 0 && srcY < height && srcX >= 0 && srcX < width)
            N_ds[destY][destX] = imageInput[src];
        else
            N_ds[destY][destX] = 0;
    }
    __syncthreads();

    int Pvalue = 0;
    int y, x;
    for (y = 0; y < maskWidth; y++)
        for (x = 0; x < maskWidth; x++)
            Pvalue += N_ds[threadIdx.y + y][threadIdx.x + x] * M[y * maskWidth + x];
    y = blockIdx.y * TILE_SIZE + threadIdx.y;
    x = blockIdx.x * TILE_SIZE + threadIdx.x;
    if (y < height && x < width)
        imageOutput[(y * width + x)] = clamp(Pvalue);
    __syncthreads();
}

//Llamado al Kernel

void convolution2DKernelCall(Mat image,unsigned char *In,unsigned char *Out,char *h_Mask,
  int Mask_Width,int Row,int Col, int op)
{
  // Variables
  int Size_of_bytes =  sizeof(unsigned char)*Row*Col*image.channels();
  int Mask_size_bytes =  sizeof(char)*(Mask_size*Mask_size);
  unsigned char *d_In, *d_Out;
  char *d_Mask;
  float Blocksize=BLOCK_SIZE;


  // Asignación de memoria en Device
  hipMalloc((void**)&d_In,Size_of_bytes);
  hipMalloc((void**)&d_Out,Size_of_bytes);
  hipMalloc((void**)&d_Mask,Mask_size_bytes);

  // Memcpy Host to device
  hipMemcpy(d_In,In,Size_of_bytes,hipMemcpyHostToDevice);
  hipMemcpy(d_Mask,h_Mask,Mask_size_bytes,hipMemcpyHostToDevice);
  hipMemcpyToSymbol(HIP_SYMBOL(M),h_Mask,Mask_size_bytes);// Using constant mem

  dim3 dimGrid(ceil(Row/Blocksize),ceil(Col/Blocksize),1);
  dim3 dimBlock(Blocksize,Blocksize,1);

  //Para seleccionar cual kernel se quiere ejecutar
  switch(op)
  {
    case 1:
    cout<<"2D convolution usando memoria Global"<<endl;
    convolution2DGlobalMemKernel<<<dimGrid,dimBlock>>>(d_In,d_Mask,d_Out,Mask_Width,Row,Col);
    break;
    case 2:
    cout<<"2D convolution usando memoria Constante"<<endl;
    convolution2DConstantMemKernel<<<dimGrid,dimBlock>>>(d_In,d_Out,Mask_Width,Row,Col);
    break;
    case 3:
    cout<<"2D convolution usando memoria Compartida"<<endl;
    convolution2DSharedMemKernel<<<dimGrid,dimBlock>>>(d_In,d_Out,Mask_Width,Row,Col);
    break;
  }

  hipDeviceSynchronize();

  // Guarda los resultados de salida
  hipMemcpy (Out,d_Out,Size_of_bytes,hipMemcpyDeviceToHost);

  // Memoria liberada Device
  hipFree(d_In);
  hipFree(d_Out);
  hipFree(d_Mask);
}

int main()
{
  //Clock variables
  clock_t start, finish;
  double elapsedParallel;
  double elapsedSequential;
  int Mask_Width =  Mask_size;
  Mat image;
  image = imread("inputs/img4.jpg",0);   // Lee el archivo, 0 significa que ya existe una imagen cargada en esacala de grises
  int op = 3;
  Size s = image.size();
  int Row = s.width;
  int Col = s.height;
  char h_Mask[] = {-1,0,1,-2,0,2,-1,0,1};

  unsigned char *img = (unsigned char*)malloc(sizeof(unsigned char)*Row*Col*image.channels());
  unsigned char *imgOut = (unsigned char*)malloc(sizeof(unsigned char)*Row*Col*image.channels());

  if( !image.data )
  {
    cout<<"Problema cargando la imágen"<<endl;
    return -1;
  }

  img = image.data;

  cout<<"Resultado serial"<<endl;
  Mat grad_x;
  start = clock();
  Sobel(image,grad_x,CV_8UC1,1,0,3,1,0,BORDER_DEFAULT);
  finish = clock();
  elapsedSequential = (((double) (finish - start)) / CLOCKS_PER_SEC );
  cout<< "El proceso secuencial tardó: " << elapsedSequential << " segundos en ejecutarse "<< endl;
  start = clock();
  convolution2DKernelCall(image,img,imgOut,h_Mask,Mask_Width,Row,Col,op);
  finish = clock();
  elapsedParallel = (((double) (finish - start)) / CLOCKS_PER_SEC );
  cout<< "El proceso paralelo tardó: " << elapsedParallel << " segundos en ejecutarse "<< endl;

  Mat gray_image;
  gray_image.create(Col,Row,CV_8UC1);
  gray_image.data = imgOut;
  imwrite("./outputs/1053823121.png",gray_image);
  return 0;
}
